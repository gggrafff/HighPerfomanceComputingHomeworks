#include "hip/hip_runtime.h"
// Чтобы в Clion было удобнее работать, добавим некоторые определения.
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__

// This is slightly mental, but gets it to properly index device function calls like __popc and whatever.
#define __HIPCC__
#include <hip/device_functions.h>

// These headers are all implicitly present when you compile CUDA with clang. Clion doesn't know that, so
// we include them explicitly to make the indexer happy. Doing this when you actually build is, obviously,
// a terrible idea :D
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif // __JETBRAINS_IDE__


#include <fstream>
#include <cassert>

#include "BMP.h"  // https://github.com/sol-prog/cpp-bmp-images

/**
 * @brief Простая сортировка элементов массива за n^2.
 * @details Эффективно только для маленьких массивов.
 * @details На маленьких массивах накладные расходы более сложных алгоритмов сортировки не всегда себя окупают.
 * @param arr Массив.
 * @param size Размер массива.
 */
__device__ void sort(uint16_t *arr, size_t size)
{
    for (int i = 0; i < size; i++) {
        for(int j = i + 1; j < size; j++) {
            if (arr[j] < arr[i]) {
                uint16_t temp = arr[j];
                arr[j] = arr[i];
                arr[i] = temp;
            }
        }
    }
}


/**
 * @brief Медианный фильтр.
 * @param d_input Массив пикселей исходного изображения размером 3 * input_width * input_height.
 * @param input_width Ширина в пикселях исходного изображения.
 * @param input_height Высота в пикселях исходного изображения.
 * @param d_output Массив пикселей отфильтрованного изображения размером 3 * input_width * input_height.
 * @param output_width Ширина в пикселях отфильтрованного изображения.
 * @param output_height Высота в пикселях отфильтрованного изображения.
 * @param filter_size Размер области для поиска медианы (квадратная область filter_size * filter_size).
 */
__global__ void filter(
        uint8_t *d_input,
        size_t input_width,
        size_t input_height,
        uint8_t *d_output,
        size_t output_width,
        size_t output_height,
        size_t filter_size)
{
    /**
     * Прочитал, что обращение к shared памяти может быть быстрее в сотни раз, чем обращение к global памяти.
     * В случаях, когда обращение к элементам производится более одного раза,
     * предварительное копирование элементов в shared память часто даёт хороший прирост производительности.
     * Решил попробовать копировать элементы из global в shared перед дальнейшей работой.
     * Dynamic shared memory допускает использование только одного массива в kernel.
     * Кроме того, необходимо использовать клетки-призраки на границах блоков.
     * Эти факторы привели к довольно страшной индексации элементов.
     */

    // Адресуемся к пикселям исходного и итогового изображений.
    const int output_center_column_no = blockIdx.x * blockDim.x + threadIdx.x;
    const int output_center_row_no = blockIdx.y * blockDim.y + threadIdx.y;
    if ((output_center_column_no < output_width) && (output_center_row_no < output_height)) {
        size_t input_center_row_no = output_center_row_no + filter_size / 2;
        size_t input_center_column_no = output_center_column_no + filter_size / 2;

        // массивы пикселей, за которые отвечает данный блок, с клетками-призраками
        extern __shared__ uint8_t pixels[];
        size_t pixels_width = blockDim.x + (filter_size / 2) * 2;
        size_t pixels_height = pixels_width;
        uint8_t* r = pixels;
        uint8_t* g = pixels + pixels_width * pixels_height;
        uint8_t* b = pixels + 2 * pixels_width * pixels_height;

        if (threadIdx.x > blockDim.x - filter_size / 2 - 1 && threadIdx.y > blockDim.y - filter_size / 2 - 1) {
            // правые нижние клетки-призраки
            size_t pixel_idx = (input_center_row_no + filter_size / 2) * input_width + input_center_column_no + filter_size / 2;
            r[(threadIdx.x + (filter_size / 2) * 2) * pixels_width + threadIdx.y + (filter_size / 2) * 2] = d_input[3 * pixel_idx + 0];
            g[(threadIdx.x + (filter_size / 2) * 2) * pixels_width + threadIdx.y + (filter_size / 2) * 2] = d_input[3 * pixel_idx + 1];
            b[(threadIdx.x + (filter_size / 2) * 2) * pixels_width + threadIdx.y + (filter_size / 2) * 2] = d_input[3 * pixel_idx + 2];
        }
        if (threadIdx.x > blockDim.x - filter_size / 2 - 1 && threadIdx.y < filter_size / 2) {
            // правые верхние клетки-призраки
            size_t pixel_idx = (input_center_row_no - filter_size / 2) * input_width + (input_center_column_no + filter_size / 2);
            r[(threadIdx.x + (filter_size / 2) * 2) * pixels_width + threadIdx.y] = d_input[3 * pixel_idx + 0];
            g[(threadIdx.x + (filter_size / 2) * 2) * pixels_width + threadIdx.y] = d_input[3 * pixel_idx + 1];
            b[(threadIdx.x + (filter_size / 2) * 2) * pixels_width + threadIdx.y] = d_input[3 * pixel_idx + 2];
        }
        if (threadIdx.x < filter_size / 2 && threadIdx.y > blockDim.y - filter_size / 2 - 1) {
            // левые нижние клетки-призраки
            size_t pixel_idx = (input_center_row_no + filter_size / 2) * input_width + (input_center_column_no - filter_size / 2);
            r[(threadIdx.x) * pixels_width + threadIdx.y + (filter_size / 2) * 2] = d_input[3 * pixel_idx + 0];
            g[(threadIdx.x) * pixels_width + threadIdx.y + (filter_size / 2) * 2] = d_input[3 * pixel_idx + 1];
            b[(threadIdx.x) * pixels_width + threadIdx.y + (filter_size / 2) * 2] = d_input[3 * pixel_idx + 2];
        }
        if (threadIdx.x < filter_size / 2 && threadIdx.y < filter_size / 2) {
            // левые верхние клетки-призраки
            size_t pixel_idx = (input_center_row_no - filter_size / 2) * input_width + (input_center_column_no - filter_size / 2);
            r[(threadIdx.x) * pixels_width + threadIdx.y] = d_input[3 * pixel_idx + 0];
            g[(threadIdx.x) * pixels_width + threadIdx.y] = d_input[3 * pixel_idx + 1];
            b[(threadIdx.x) * pixels_width + threadIdx.y] = d_input[3 * pixel_idx + 2];
        }
        if (threadIdx.x < filter_size / 2) {  // левые клетки-призраки
            size_t pixel_idx = input_center_row_no * input_width + (input_center_column_no - filter_size / 2);
            r[(threadIdx.x) * pixels_width + threadIdx.y + filter_size / 2] = d_input[3 * pixel_idx + 0];
            g[(threadIdx.x) * pixels_width + threadIdx.y + filter_size / 2] = d_input[3 * pixel_idx + 1];
            b[(threadIdx.x) * pixels_width + threadIdx.y + filter_size / 2] = d_input[3 * pixel_idx + 2];
        }
        if (threadIdx.x > blockDim.x - filter_size / 2 - 1) {  // правые клетки-призраки
            size_t pixel_idx = input_center_row_no * input_width + (input_center_column_no + filter_size / 2);
            r[(threadIdx.x + (filter_size / 2) * 2) * pixels_width + threadIdx.y + filter_size / 2] = d_input[3 * pixel_idx + 0];
            g[(threadIdx.x + (filter_size / 2) * 2) * pixels_width + threadIdx.y + filter_size / 2] = d_input[3 * pixel_idx + 1];
            b[(threadIdx.x + (filter_size / 2) * 2) * pixels_width + threadIdx.y + filter_size / 2] = d_input[3 * pixel_idx + 2];
        }
        if (threadIdx.y < filter_size / 2) {  // верхние клетки-призраки
            size_t pixel_idx = (input_center_row_no - filter_size / 2) * input_width + input_center_column_no;
            r[(threadIdx.x + filter_size / 2) * pixels_width + threadIdx.y] = d_input[3 * pixel_idx + 0];
            g[(threadIdx.x + filter_size / 2) * pixels_width + threadIdx.y] = d_input[3 * pixel_idx + 1];
            b[(threadIdx.x + filter_size / 2) * pixels_width + threadIdx.y] = d_input[3 * pixel_idx + 2];
        }
        if (threadIdx.y > blockDim.y - filter_size / 2 - 1) {  // нижние клетки-призраки
            size_t pixel_idx = (input_center_row_no + filter_size / 2) * input_width + input_center_column_no;
            r[(threadIdx.x + filter_size / 2) * pixels_width + threadIdx.y + (filter_size / 2) * 2] = d_input[3 * pixel_idx + 0];
            g[(threadIdx.x + filter_size / 2) * pixels_width + threadIdx.y + (filter_size / 2) * 2] = d_input[3 * pixel_idx + 1];
            b[(threadIdx.x + filter_size / 2) * pixels_width + threadIdx.y + (filter_size / 2) * 2] = d_input[3 * pixel_idx + 2];
        }

        // клетки не призраки
        size_t pixel_idx = input_center_row_no * input_width + input_center_column_no;
        r[(threadIdx.x + filter_size / 2) * pixels_width + threadIdx.y + filter_size / 2] = d_input[3 * pixel_idx + 0];
        g[(threadIdx.x + filter_size / 2) * pixels_width + threadIdx.y + filter_size / 2] = d_input[3 * pixel_idx + 1];
        b[(threadIdx.x + filter_size / 2) * pixels_width + threadIdx.y + filter_size / 2] = d_input[3 * pixel_idx + 2];

        // Дожидаемся, когда всё будет скопировано в shared память.
        __syncthreads();

        // Создаём в каждой нити массивы, которые будем сортировать для поиска медианы.
        uint16_t* window_r = new uint16_t[filter_size * filter_size];
        uint16_t* window_g = new uint16_t[filter_size * filter_size];
        uint16_t* window_b = new uint16_t[filter_size * filter_size];

        size_t window_idx = 0;

        // Заполняем массивы.
        for (int i = 0; i < filter_size; i ++)
        {
            for (int j = 0; j < filter_size; j++)
            {
                window_r[window_idx] = r[(threadIdx.x + i) * pixels_width + threadIdx.y + j];
                window_g[window_idx] = g[(threadIdx.x + i) * pixels_width + threadIdx.y + j];
                window_b[window_idx] = b[(threadIdx.x + i) * pixels_width + threadIdx.y + j];
                window_idx += 1;
            }
        }

        // Сортируем массивы.
        sort(window_r, filter_size * filter_size);
        sort(window_g, filter_size * filter_size);
        sort(window_b, filter_size * filter_size);

        // Получаем значения медиан.
        uint8_t r_median = static_cast<uint8_t>((window_r[filter_size * filter_size / 2] + window_r[(filter_size * filter_size + 1) / 2]) / 2);
        uint8_t g_median = static_cast<uint8_t>((window_g[filter_size * filter_size / 2] + window_g[(filter_size * filter_size + 1) / 2]) / 2);
        uint8_t b_median = static_cast<uint8_t>((window_b[filter_size * filter_size / 2] + window_b[(filter_size * filter_size + 1) / 2]) / 2);

        // Записываем результат.
        size_t output_center_idx = output_center_row_no * output_width + output_center_column_no;
        d_output[3 * output_center_idx + 0] = r_median;
        d_output[3 * output_center_idx + 1] = g_median;
        d_output[3 * output_center_idx + 2] = b_median;

        // Чистим память.
        delete[] window_r;
        delete[] window_g;
        delete[] window_b;
    }
}

int main(int argc, char *argv[])
{
    // Введение в CUDA: https://developer.nvidia.com/blog/easy-introduction-cuda-c-and-c/

    assert(argc == 4);  // SimpleFilter input.bmp output.bmp filter_size

    // Задаём размер фильтра.
    const size_t filter_size = std::stoul(argv[3]);

    // Читаем исходное изображение.
    BMP input_image(argv[1]);
    assert(input_image.data.size() == 3 * input_image.bmp_info_header.width * input_image.bmp_info_header.height);

    // Создаём объект отфильтрованного изображения.
    BMP output_image(input_image.bmp_info_header.width - filter_size + 1, input_image.bmp_info_header.height - filter_size + 1, false);
    assert(output_image.data.size() == 3 * output_image.bmp_info_header.width * output_image.bmp_info_header.height);

    // Выделяем память на видеокарте и копируем туда исходное изображение.
    uint8_t* d_input, *d_output;
    hipMalloc(&d_input, input_image.data.size());
    hipMalloc(&d_output, output_image.data.size());
    hipMemcpy(d_input, input_image.data.data(), input_image.data.size(), hipMemcpyHostToDevice);

    // Запускаем фильтрацию.
    // Про индексацию:
    // https://medium.com/@erangadulshan.14/1d-2d-and-3d-thread-allocation-for-loops-in-cuda-e0f908537a52
    // https://www.researchgate.net/post/The_optimal_number_of_threads_per_block_in_CUDA_programming
    dim3 dimBlock(16, 16);
    dim3 dimGrid((output_image.bmp_info_header.width + 15) / 16, (output_image.bmp_info_header.height + 15) / 16);
    // Про динамическую общую память: https://stackoverflow.com/questions/24942073/dynamic-shared-memory-in-cuda
    const size_t shared_memory_size = (dimBlock.x + (filter_size / 2) * 2) * (dimBlock.x + (filter_size / 2) * 2) * 3;
    filter<<<dimGrid, dimBlock, shared_memory_size>>>(
            d_input,
            input_image.bmp_info_header.width,
            input_image.bmp_info_header.height,
            d_output,
            output_image.bmp_info_header.width,
            output_image.bmp_info_header.height,
            filter_size);

    // Забираем результат с видеокарты.
    hipMemcpy(const_cast<uint8_t*>(output_image.data.data()), d_output, output_image.data.size(), hipMemcpyDeviceToHost);

    // Сохраняем.
    output_image.write(argv[2]);

    // Очищаем память.
    hipFree(d_input);
    hipFree(d_output);
}