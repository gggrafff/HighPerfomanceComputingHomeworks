#include "hip/hip_runtime.h"
// Чтобы в Clion было удобнее работать, добавим некоторые определения.
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__

// This is slightly mental, but gets it to properly index device function calls like __popc and whatever.
#define __HIPCC__
#include <hip/device_functions.h>

// These headers are all implicitly present when you compile CUDA with clang. Clion doesn't know that, so
// we include them explicitly to make the indexer happy. Doing this when you actually build is, obviously,
// a terrible idea :D
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif // __JETBRAINS_IDE__


#include <fstream>
#include <cassert>

#include "BMP.h"  // https://github.com/sol-prog/cpp-bmp-images


/**
 * @brief Получение глабольного индекса потока при 1d адресации блоков и 1d адресации потоков внутри блока.
 * @details https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
 * @return Глобальный индекс нити.
 */
__device__ int getGlobalIdx_1D_1D() {
    return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void histogram(
        uint8_t *d_input,
        size_t input_width,
        size_t input_height,
        int *d_r_hist,
        int *d_g_hist,
        int *d_b_hist,
        size_t width_of_intervals)
{
    // Адресуемся к пикселям исходного и итогового изображений.
    const int input_center_idx = getGlobalIdx_1D_1D();
    if (input_center_idx < input_width * input_height) {
        // Подсчитываем количество значений в интервалах.
        atomicAdd(d_r_hist + d_input[3 * input_center_idx + 0] / width_of_intervals, 1);
        atomicAdd(d_g_hist + d_input[3 * input_center_idx + 1] / width_of_intervals, 1);
        atomicAdd(d_b_hist + d_input[3 * input_center_idx + 2] / width_of_intervals, 1);
    }
}

int main(int argc, char *argv[])
{
    // Введение в CUDA: https://developer.nvidia.com/blog/easy-introduction-cuda-c-and-c/

    assert(argc == 3);  // Histogram input.bmp count_of_intervals

    // Задаём размер фильтра.
    const size_t count_of_intervals = std::stoul(argv[2]);
    assert(count_of_intervals <= 256);
    assert(256 % count_of_intervals == 0);
    const size_t width_of_intervals = 256 / count_of_intervals;

    // Читаем исходное изображение.
    BMP input_image(argv[1]);
    assert(input_image.data.size() == 3 * input_image.bmp_info_header.width * input_image.bmp_info_header.height);

    // Выделяем память на видеокарте и копируем туда исходное изображение.
    uint8_t *d_input;
    int *d_r_hist, *d_g_hist, *d_b_hist;
    hipMalloc(&d_input, input_image.data.size());
    hipMemcpy(d_input, input_image.data.data(), input_image.data.size(), hipMemcpyHostToDevice);
    hipMalloc(&d_r_hist, sizeof(int) * count_of_intervals);
    hipMalloc(&d_g_hist, sizeof(int) * count_of_intervals);
    hipMalloc(&d_b_hist, sizeof(int) * count_of_intervals);

    // Запускаем подсчёт гистограммы.
    // Про индексацию:
    // https://medium.com/@erangadulshan.14/1d-2d-and-3d-thread-allocation-for-loops-in-cuda-e0f908537a52
    // https://www.researchgate.net/post/The_optimal_number_of_threads_per_block_in_CUDA_programming
    histogram<<<(input_image.data.size() + 255) / 256, 256>>>(
            d_input,
            input_image.bmp_info_header.width,
            input_image.bmp_info_header.height,
            d_r_hist,
            d_g_hist,
            d_b_hist,
            width_of_intervals);

    // Забираем результат с видеокарты.
    int *r_hist, *g_hist, *b_hist;
    r_hist = new int[count_of_intervals];
    g_hist = new int[count_of_intervals];
    b_hist = new int[count_of_intervals];
    hipMemcpy(r_hist, d_r_hist, count_of_intervals * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(g_hist, d_g_hist, count_of_intervals * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b_hist, d_b_hist, count_of_intervals * sizeof(int), hipMemcpyDeviceToHost);

    // Сохраняем.
    std::ofstream r_hist_out("r_hist.txt");
    std::ofstream g_hist_out("g_hist.txt");
    std::ofstream b_hist_out("b_hist.txt");
    for(size_t i = 0; i < count_of_intervals; ++i) {
        r_hist_out << r_hist[i] << " ";
        g_hist_out << g_hist[i] << " ";
        b_hist_out << b_hist[i] << " ";
    }
    r_hist_out.close();
    g_hist_out.close();
    b_hist_out.close();

    // Очищаем память.
    hipFree(d_input);
    hipFree(d_r_hist);
    hipFree(d_g_hist);
    hipFree(d_b_hist);

    delete[] r_hist;
    delete[] g_hist;
    delete[] b_hist;
}