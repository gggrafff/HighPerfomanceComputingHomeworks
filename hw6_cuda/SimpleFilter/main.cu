#include "hip/hip_runtime.h"
// Чтобы в Clion было удобнее работать, добавим некоторые определения.
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__

// This is slightly mental, but gets it to properly index device function calls like __popc and whatever.
#define __HIPCC__
#include <hip/device_functions.h>

// These headers are all implicitly present when you compile CUDA with clang. Clion doesn't know that, so
// we include them explicitly to make the indexer happy. Doing this when you actually build is, obviously,
// a terrible idea :D
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif // __JETBRAINS_IDE__


#include <fstream>
#include <cassert>

#include "BMP.h"  // https://github.com/sol-prog/cpp-bmp-images

/**
 * @brief Получение глабольного индекса потока при 2d адресации блоков и 2d адресации потоков внутри блока.
 * @details https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
 * @return Глобальный индекс нити.
 */
__device__ size_t getGlobalIdx_2D_2D() {
    size_t blockId = blockIdx.x + blockIdx.y * gridDim.x;
    size_t threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

/**
 * @brief Получение глабольного индекса потока при 1d адресации блоков и 1d адресации потоков внутри блока.
 * @details https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
 * @return Глобальный индекс нити.
 */
__device__ int getGlobalIdx_1D_1D() {
    return blockIdx.x *blockDim.x + threadIdx.x;
}

/**
 * @brief Простой фильтр, ищет взвешенное среднее значение пикселей.
 * @param d_input Массив пикселей исходного изображения размером 3 * input_width * input_height.
 * @param input_width Ширина в пикселях исходного изображения.
 * @param input_height Высота в пикселях исходного изображения.
 * @param d_output Массив пикселей отфильтрованного изображения размером 3 * input_width * input_height.
 * @param output_width Ширина в пикселях отфильтрованного изображения.
 * @param output_height Высота в пикселях отфильтрованного изображения.
 * @param filter_size Рамзер шаблона фильтра.
 * @param d_pattern Веса фильтра. Массив размером filter_size * filter_size.
 * @param pattern_sum Сумма весов фильтра.
 */
__global__ void filter(
        uint8_t *d_input,
        size_t input_width,
        size_t input_height,
        uint8_t *d_output,
        size_t output_width,
        size_t output_height,
        size_t filter_size,
        uint8_t* d_pattern,
        uint16_t pattern_sum)
{
    // Адресуемся к пикселям исходного и итогового изображений.
    auto output_center_idx = getGlobalIdx_1D_1D();
    if (output_center_idx < output_width * output_height) {
        size_t output_center_row_no = output_center_idx / output_width;
        size_t output_center_column_no = output_center_idx % output_width;
        size_t input_center_row_no = output_center_row_no + filter_size / 2;
        size_t input_center_column_no = output_center_column_no + filter_size / 2;

        // Средние занчения пикселей сохраним в переменные.
        uint8_t r = 0;
        uint8_t g = 0;
        uint8_t b = 0;

        // Усредняем с весами.
        for(auto i = 0; i < filter_size; ++i){
            for(auto j = 0; j < filter_size; ++j){
                size_t pixel_idx = (j + input_center_row_no - filter_size / 2) * input_width + (i + input_center_column_no - filter_size / 2);
                r += std::abs(static_cast<int16_t>(d_input[3 * pixel_idx + 0]) * d_pattern[i * filter_size + j] / pattern_sum);
                g += std::abs(static_cast<int16_t>(d_input[3 * pixel_idx + 1]) * d_pattern[i * filter_size + j] / pattern_sum);
                b += std::abs(static_cast<int16_t>(d_input[3 * pixel_idx + 2]) * d_pattern[i * filter_size + j] / pattern_sum);
            }
        }

        // Записываем результат.
        d_output[3 * output_center_idx + 0] = r;
        d_output[3 * output_center_idx + 1] = g;
        d_output[3 * output_center_idx + 2] = b;
    }
}

int main(int argc, char *argv[])
{
    // Введение в CUDA: https://developer.nvidia.com/blog/easy-introduction-cuda-c-and-c/

    assert(argc == 5);  // SimpleFilter input.bmp output.bmp filter_size pattern.txt

    // Задаём размер фильтра.
    const size_t filter_size = std::stoul(argv[3]);
    assert(filter_size % 2 == 1);

    // Читаем исходное изображение.
    BMP input_image(argv[1]);
    assert(input_image.data.size() == 3 * input_image.bmp_info_header.width * input_image.bmp_info_header.height);

    // Создаём объект отфильтрованного изображения.
    BMP output_image(input_image.bmp_info_header.width - filter_size + 1, input_image.bmp_info_header.height - filter_size + 1, false);
    assert(output_image.data.size() == 3 * output_image.bmp_info_header.width * output_image.bmp_info_header.height);

    // Выделяем память на видеокарте и копируем туда исходное изображение.
    uint8_t* d_input, *d_output;
    hipMalloc(&d_input, input_image.data.size());
    hipMalloc(&d_output, output_image.data.size());
    hipMemcpy(d_input, input_image.data.data(), input_image.data.size(), hipMemcpyHostToDevice);

    // Читаем фильтр из файла и копируем в память видеокарты.
    uint8_t* pattern;
    uint8_t* d_pattern;
    uint16_t pattern_sum = 0;
    pattern = new uint8_t[filter_size * filter_size];
    std::ifstream pattern_file(argv[4]);
    for(auto i = 0; i < filter_size; ++i) {
        for(auto j = 0; j < filter_size; ++j) {
            uint16_t pattern_element;
            pattern_file >> pattern_element;
            pattern[i * filter_size + j] = pattern_element;
            pattern_sum += pattern_element;
            std::cout << pattern_element << " ";
        }
        std::cout << "\n";
    }
    pattern_file.close();
    if (pattern_sum == 0) {
        pattern_sum = 1;
        std::cout << "pattern_sum is zero\n";
    }
    std::cout << "pattern_sum == " << pattern_sum << "\n";
    hipMalloc(&d_pattern, filter_size * filter_size);
    hipMemcpy(d_pattern, pattern, filter_size * filter_size, hipMemcpyHostToDevice);

    // Запускаем фильтрацию.
    // Про индексацию:
    // https://medium.com/@erangadulshan.14/1d-2d-and-3d-thread-allocation-for-loops-in-cuda-e0f908537a52
    // https://www.researchgate.net/post/The_optimal_number_of_threads_per_block_in_CUDA_programming
    filter<<<(output_image.data.size() + 255) / 256, 256>>>(
            d_input,
            input_image.bmp_info_header.width,
            input_image.bmp_info_header.height,
            d_output,
            output_image.bmp_info_header.width,
            output_image.bmp_info_header.height,
            filter_size,
            d_pattern,
            pattern_sum);

    // Забираем результат с видеокарты.
    hipMemcpy(const_cast<uint8_t*>(output_image.data.data()), d_output, output_image.data.size(), hipMemcpyDeviceToHost);

    // Сохраняем.
    output_image.write(argv[2]);

    // Очищаем память.
    hipFree(d_input);
    hipFree(d_output);
    delete[] pattern;
    hipFree(d_pattern);
}