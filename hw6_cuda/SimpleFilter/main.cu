#include "hip/hip_runtime.h"
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__

// This is slightly mental, but gets it to properly index device function calls like __popc and whatever.
#define __HIPCC__
#include <hip/device_functions.h>

// These headers are all implicitly present when you compile CUDA with clang. Clion doesn't know that, so
// we include them explicitly to make the indexer happy. Doing this when you actually build is, obviously,
// a terrible idea :D
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif // __JETBRAINS_IDE__


#include <fstream>
#include <cassert>

#include "BMP.h"  // https://github.com/sol-prog/cpp-bmp-images

// https://cs.calvin.edu/courses/cs/374/CUDA/CUDA-Thread-Indexing-Cheatsheet.pdf
__device__ size_t getGlobalIdx_2D_2D() {
    size_t blockId = blockIdx.x + blockIdx.y * gridDim.x;
    size_t threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__device__ int getGlobalIdx_1D_1D() {
    return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void filter(
        uint8_t *d_input,
        size_t input_width,
        size_t input_height,
        uint8_t *d_output,
        size_t output_width,
        size_t output_height,
        size_t filter_size,
        uint8_t* d_pattern,
        uint16_t pattern_sum)
{
    auto output_center_idx = getGlobalIdx_1D_1D();
    if (output_center_idx < output_width * output_height) {
        size_t output_center_row_no = output_center_idx / output_width;
        size_t output_center_column_no = output_center_idx % output_width;
        size_t input_center_row_no = output_center_row_no + filter_size / 2;
        size_t input_center_column_no = output_center_column_no + filter_size / 2;
        // size_t input_center_idx = input_center_row_no * input_width + input_center_column_no;

        uint8_t r = 0;
        uint8_t g = 0;
        uint8_t b = 0;

        for(auto i = 0; i < filter_size; ++i){
            for(auto j = 0; j < filter_size; ++j){
                size_t pixel_idx = (j + input_center_row_no - filter_size / 2) * input_width + (i + input_center_column_no - filter_size / 2);
                r += std::abs(static_cast<int16_t>(d_input[3 * pixel_idx + 0]) * d_pattern[i * filter_size + j] / pattern_sum);
                g += std::abs(static_cast<int16_t>(d_input[3 * pixel_idx + 1]) * d_pattern[i * filter_size + j] / pattern_sum);
                b += std::abs(static_cast<int16_t>(d_input[3 * pixel_idx + 2]) * d_pattern[i * filter_size + j] / pattern_sum);
            }
        }

        assert((r < 256) && (r >= 0));
        assert((g < 256) && (g >= 0));
        assert((b < 256) && (b >= 0));

        d_output[3 * output_center_idx + 0] = r;
        d_output[3 * output_center_idx + 1] = g;
        d_output[3 * output_center_idx + 2] = b;
    }
}

int main(void)
{
    const size_t filter_size = 5;
    assert(filter_size % 2 == 1);

    BMP input_image("image_24b.bmp");
    assert(input_image.data.size() == 3 * input_image.bmp_info_header.width * input_image.bmp_info_header.height);

    BMP output_image(input_image.bmp_info_header.width - filter_size + 1, input_image.bmp_info_header.height - filter_size + 1, false);
    assert(output_image.data.size() == 3 * output_image.bmp_info_header.width * output_image.bmp_info_header.height);

    // https://developer.nvidia.com/blog/easy-introduction-cuda-c-and-c/
    uint8_t* d_input, *d_output;
    hipMalloc(&d_input, input_image.data.size());
    hipMalloc(&d_output, output_image.data.size());
    hipMemcpy(d_input, input_image.data.data(), input_image.data.size(), hipMemcpyHostToDevice);

    uint8_t* pattern;
    uint8_t* d_pattern;
    uint16_t pattern_sum = 0;
    pattern = new uint8_t[filter_size * filter_size];
    std::ifstream pattern_file("pattern2.txt");
    for(auto i = 0; i < filter_size; ++i) {
        for(auto j = 0; j < filter_size; ++j) {
            uint16_t pattern_element;
            pattern_file >> pattern_element;
            pattern[i * filter_size + j] = pattern_element;
            pattern_sum += pattern_element;
            std::cout << pattern_element << " ";
        }
        std::cout << "\n";
    }
    pattern_file.close();
    if (pattern_sum == 0) {
        pattern_sum = 1;
        std::cout << "pattern_sum is zero\n";
    }
    std::cout << "pattern_sum == " << pattern_sum << "\n";
    hipMalloc(&d_pattern, filter_size * filter_size);
    hipMemcpy(d_pattern, pattern, filter_size * filter_size, hipMemcpyHostToDevice);

    // https://medium.com/@erangadulshan.14/1d-2d-and-3d-thread-allocation-for-loops-in-cuda-e0f908537a52
    //dim3 grid((output_image.bmp_info_header.width + 15) / 16, (output_image.bmp_info_header.height + 15) / 16);
    //dim3 block(16, 16);
    //filter<<<grid, block>>>(
    // https://www.researchgate.net/post/The_optimal_number_of_threads_per_block_in_CUDA_programming
    filter<<<(output_image.data.size() + 255) / 256, 256>>>(
            d_input,
            input_image.bmp_info_header.width,
            input_image.bmp_info_header.height,
            d_output,
            output_image.bmp_info_header.width,
            output_image.bmp_info_header.height,
            filter_size,
            d_pattern,
            pattern_sum);

    hipMemcpy(const_cast<uint8_t*>(output_image.data.data()), d_output, output_image.data.size(), hipMemcpyDeviceToHost);

    output_image.write("filtered_24b_p2.bmp");

    hipFree(d_input);
    hipFree(d_output);
    delete[] pattern;
    hipFree(d_pattern);
}